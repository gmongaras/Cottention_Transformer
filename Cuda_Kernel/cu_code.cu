#include "hip/hip_runtime.h"
// #include <hip/hip_runtime.h>
// #include <hipblas.h>
// #include <torch/extension.h>

// // template<typename T>
// // void compute_and_contract_cuda_impl(const T* A, const T* B, const T* C, T* output, int N, int H, int S, int D);

// // // Define the wrapper function to call the kernel
// // void compute_and_contract_cuda(const torch::Tensor& A, const torch::Tensor& B, const torch::Tensor& C, torch::Tensor& output) {
// //     // Extract tensor dimensions and call the template function
// //     auto N = A.size(0);
// //     auto H = A.size(1);
// //     auto S = A.size(2);
// //     auto D = A.size(3);

// //     AT_DISPATCH_FLOATING_TYPES(A.type(), "compute_and_contract_cuda", ([&] {
// //         compute_and_contract_cuda_impl<scalar_t>(
// //             A.data_ptr<scalar_t>(),
// //             B.data_ptr<scalar_t>(),
// //             C.data_ptr<scalar_t>(),
// //             output.data_ptr<scalar_t>(),
// //             N, H, S, D);
// //     }));
// // }

// template<typename T>
// __global__ void compute_and_contract_kernel(
//     const T* __restrict__ A, const T* __restrict__ B, const T* __restrict__ C,
//     T* __restrict__ output, int N, int H, int S, int D) {
//     extern __shared__ T sharedMem[];

//     int b = blockIdx.x; // Batch index
//     int h = blockIdx.y; // Head index
//     int s = blockIdx.z; // Sequence index
//     int d = threadIdx.x; // Dimension index

//     if (b < N && h < H && s < S && d < D) {
//         // Compute cumulative product directly
//         T cumProd = 0;
//         for (int k = 0; k <= d; ++k) {
//             int idxA = b * H * S * D + h * S * D + s * D + k;
//             int idxB = b * H * S * D + h * S * D + k * S + s; // Transpose S and D for B
//             cumProd += A[idxA] * B[idxB];
//         }

//         // Store cumulative product in shared memory for reduction
//         sharedMem[threadIdx.x] = cumProd;
//         __syncthreads();

//         // Reduction within a block to compute the final output
//         // Assuming D is a power of 2 for simplicity. For non-power of 2, additional handling is needed.
//         for (int stride = D / 2; stride > 0; stride >>= 1) {
//             if (d < stride) {
//                 sharedMem[d] += sharedMem[d + stride];
//             }
//             __syncthreads();
//         }

//         // Use the result of reduction to compute the final tensor contraction with C
//         if (d == 0) {
//             int idxC = b * H * S * D + h * S * D + s * D; // Index for C
//             int idxOutput = b * H * S * D + h * S * D + s * D; // Index for output
//             output[idxOutput] = sharedMem[0] * C[idxC];
//         }
//     }
// }

// template<typename T>
// void compute_and_contract(
//     const T* A, const T* B, const T* C, T* output,
//     int N, int H, int S, int D,
//     hipStream_t stream = 0) {
//     dim3 grid(N, H, S);
//     int threads = D;
//     int sharedMemSize = D * sizeof(T);
//     compute_and_contract_kernel<<<grid, threads, sharedMemSize, stream>>>(A, B, C, output, N, H, S, D);
// }


#include <hip/hip_runtime.h> // For hipMemcpy and hipFree
#include <torch/torch.h>
// #include <torch/extension.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <iostream>
#include <chrono>






// For debugging
#include <fstream>
template<typename T>
void writeTensorToFile(const std::string& filename, const T* tensorData, const std::vector<int>& shape) {
    std::ofstream file(filename, std::ios::binary | std::ios::out);

    // Write the shape
    int dimensions = shape.size();
    file.write(reinterpret_cast<const char*>(&dimensions), sizeof(dimensions));
    for (int dim : shape) {
        file.write(reinterpret_cast<const char*>(&dim), sizeof(dim));
    }

    // Get the number of elements in the tensor
    int numElements = 1;
    for (int dim : shape) numElements *= dim;

    // Allocate host memory to copy the tensor data
    size_t numBytes = numElements * sizeof(T);
    T* hostData = new T[numElements];

    // Copy the tensor data to host
    hipMemcpy(hostData, tensorData, numBytes, hipMemcpyDeviceToHost);

    // Write the tensor data to file
    file.write(reinterpret_cast<const char*>(hostData), sizeof(T) * numElements);

    // Close the file and free the host memory
    file.close();
    free(hostData);
}







// Used to do (K.unsqueeze(-1)*V.unsqueeze(-2))
// template<typename T>
// __global__ void compute_outer_product_kernel(
//     const T* K, const T* V, T* KV,
//     int N, int H, int S, int d_K, int d_V) {
//     int n = blockIdx.x; // Batch index
//     int h = blockIdx.y; // Head index
//     int s = blockIdx.z; // Sequence index
//     int d_k = threadIdx.x / d_V; // Dimension index for K
//     int d_v = threadIdx.x % d_V; // Dimension index for V

//     // Ensure we are within bounds since d_k and d_v are derived from threadIdx.x
//     if (d_k < d_K && d_v < d_V) {
//         // Compute linear indices for K and V
//         int indexK = ((n * H + h) * S + s) * d_K + d_k;
//         int indexV = ((n * H + h) * S + s) * d_V + d_v;

//         // Compute the index for KV
//         int indexKV = ((((n * H + h) * S + s) * d_K) + d_k) * d_V + d_v;

//         // Perform the multiplication
//         KV[indexKV] = K[indexK] * V[indexV];
//     }
// }
// template<typename T>
// void compute_outer_product(
//     const T* K, const T* V, T* KV,
//     int N, int H, int S, int d_K, int d_V,
//     hipStream_t stream = 0) {
//     // Calculate the number of blocks and threads for the outer product kernel
//     dim3 grid(N, H, S);
//     int threadsPerBlock = d_K * d_V; // This might need adjustment based on hardware limits

//     // Ensure we do not exceed the maximum number of threads per block
//     if (threadsPerBlock > 1024) {
//         // Handle error or adjust grid and block dimensions
//         std::cerr << "Error: Number of threads per block exceeds hardware limit." << std::endl;
//         return;
//     }

//     writeTensorToFile("K.bin", K, {N, H, S, d_K, 1});
//     writeTensorToFile("V.bin", V, {N, H, S, 1, d_V});

//     // Launch the outer product kernel
//     compute_outer_product_kernel<T><<<grid, threadsPerBlock, 0, stream>>>(K, V, KV, N, H, S, d_K, d_V);

//     writeTensorToFile("KV.bin", KV, {N, H, S, d_K, d_V});

//     return;
// }



template<typename T>
__global__ void compute_outer_product_kernel(
    const T* K, const T* V, T* KV,
    int N, int H, int S, int d_K, int d_V) {
    int n = blockIdx.x; // Batch index
    int h = blockIdx.y; // Head index
    int s = blockIdx.z; // Sequence index

    int threadsPerBlock = blockDim.x;
    int numElements = d_K * d_V;
    int elementsPerThread = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    int startIdx = threadIdx.x * elementsPerThread;
    int endIdx = min(startIdx + elementsPerThread, numElements);

    for (int idx = startIdx; idx < endIdx; ++idx) {
        int d_k = idx / d_V; // Dimension index for K
        int d_v = idx % d_V; // Dimension index for V

        if (d_k < d_K && d_v < d_V) {
            int indexK = ((n * H + h) * S + s) * d_K + d_k;
            int indexV = ((n * H + h) * S + s) * d_V + d_v;
            int indexKV = ((((n * H + h) * S + s) * d_K) + d_k) * d_V + d_v;

            KV[indexKV] = K[indexK] * V[indexV];
        }
    }
}
template<typename T>
void compute_outer_product(
    const T* K, const T* V, T* KV,
    int N, int H, int S, int d_K, int d_V,
    hipStream_t stream = 0) {
    // Fixed number of threads per block
    int threadsPerBlock = 1024;

    dim3 grid(N, H, S);

    // Launch the outer product kernel
    compute_outer_product_kernel<T><<<grid, threadsPerBlock, 0, stream>>>(K, V, KV, N, H, S, d_K, d_V);

    // Ensure kernel execution completes before copying data
    hipDeviceSynchronize();

    writeTensorToFile("K.bin", K, {N, H, S, d_K, 1});
    writeTensorToFile("V.bin", V, {N, H, S, 1, d_V});
    writeTensorToFile("KV.bin", KV, {N, H, S, d_K, d_V});

    // Kill the program
    std::exit(0);

    return;
}





// template<typename T>
// void compute_outer_product(
//     const T* K, const T* V, T* KV,
//     int N, int H, int S, int D,
//     hipStream_t stream = 0) {
//     // Unsqueeze K along the last dimension and V along the second-to-last dimension
//     K = K.unsqueeze(-1);
//     V = V.unsqueeze(-2);

//     // Calculate the number of blocks for the outer product kernel
//     dim3 grid(N, H, S);
//     int threads = D;

//     // Call the outer product kernel
//     compute_outer_product_kernel<<<grid, threads, 0, stream>>>(K, V, KV, N, H, S, D);
// }




// Used to do (K.unsqueeze(-1)*V.unsqueeze(-2)).cumsum(2)
template<typename T>
__global__ void compute_cumsum_over_S_kernel(
    T* __restrict__ KV, // KV is both input and output
    int N, int H, int S, int D) {
    int b = blockIdx.x; // Batch index
    int h = blockIdx.y; // Head index
    int d1 = blockIdx.z / D; // Dimension index for the first D
    int d2 = blockIdx.z % D; // Dimension index for the second D
    int s = threadIdx.x; // Sequence index, used for cumsum

    if (b < N && h < H && d1 < D && d2 < D && s < S) {
        int baseIdx = b * H * S * D * D + h * S * D * D + d1 * D * D + d2;
        T sum = 0;
        for (int seq = 0; seq <= s; ++seq) {
            int idx = baseIdx + seq * D * D;
            sum += KV[idx];
            KV[idx] = sum; // Store the cumsum back into KV
        }
    }
}






// Compute the product of VK with Q: torch.einsum("bsD,bsdD->bsd", Q, VK)
template<typename T>
__global__ void matrix_vector_multiply_sum_kernel(
    const T* __restrict__ Q, const T* __restrict__ VK, T* __restrict__ output,
    int N, int H, int S, int D) {
    int b = blockIdx.x; // Batch index
    int h = blockIdx.y; // Head index
    int s = blockIdx.z; // Sequence index
    int d = threadIdx.x; // Dimension index in the output and Q

    if (b < N && h < H && s < S && d < D) {
        T sum = 0;
        int idxQ = b * H * S * D + h * S * D + s * D + d;
        int baseIdxVK = b * H * S * D * D + h * S * D * D + s * D * D + d;
        
        for (int i = 0; i < D; ++i) {
            int idxVK = baseIdxVK + i * D; // Move across the last D dimension in VK
            sum += Q[idxQ] * VK[idxVK];
        }
        int idxOutput = b * H * S * D + h * S * D + s * D + d;
        output[idxOutput] = sum;
    }
}







// Wrapper function to orchestrate the computation
template<typename T>
void compute_and_contract(
    const T* Q, const T* K, const T* V, T* output,
    int N, int H, int S, int D,
    hipStream_t stream = 0) {
    T* KV;
    hipMalloc(&KV, N * H * S * D * D * sizeof(T)); // Allocate space for KV

    // Compute the outer product KV = K.unsqueeze(-1) * V.unsqueeze(-2)
    compute_outer_product(K, V, KV, N, H, S, D, D, stream);


    // Compute the cumsum over S for KV: (K.unsqueeze(-1) * V.unsqueeze(-2)).cumsum(2)
    // Calculate the number of blocks for the cumsum kernel
    dim3 gridCumsum(N, H, D * D);
    int threadsCumsum = S;
    // Make sure S does not exceed the maximum number of threads per block
    // If S is larger, you'll need to adjust the strategy for cumsum calculation
    compute_cumsum_over_S_kernel<<<gridCumsum, threadsCumsum, 0, stream>>>(KV, N, H, S, D);



    // Compute the product of VK with Q: torch.einsum("bsD,bsdD->bsd", Q, VK)
    // Calculate matrix-vector multiplication and summation
    dim3 gridMulti(N, H, S);
    int threadsMulti = D;
    matrix_vector_multiply_sum_kernel<<<gridMulti, threadsMulti, 0, stream>>>(Q, KV, output, N, H, S, D);



    hipFree(KV);
}



// // CUDA forward declarations
// void compute_and_contract_cuda(const torch::Tensor& A, const torch::Tensor& B, const torch::Tensor& C, torch::Tensor& output);
// void compute_and_contract(const torch::Tensor& A, const torch::Tensor& B, const torch::Tensor& C, torch::Tensor& output);

// C++ interface
void compute_and_contract_call(const torch::Tensor& Q, const torch::Tensor& K_orig, const torch::Tensor& V_orig, torch::Tensor& output) {
    // Check tensor requirements, e.g., dtype, device, etc.
    TORCH_CHECK(Q.device().is_cuda(), "Q must be a CUDA tensor");
    TORCH_CHECK(K_orig.device().is_cuda(), "K must be a CUDA tensor");
    TORCH_CHECK(V_orig.device().is_cuda(), "V must be a CUDA tensor");
    TORCH_CHECK(output.device().is_cuda(), "output must be a CUDA tensor");

    // Get tensor dimensions
    auto N = Q.size(0);
    auto H = Q.size(1);
    auto S = Q.size(2);
    auto D = Q.size(3);

    // Unsqueeze K along the last dimension and V along the second-to-last dimension
    auto K = K_orig.unsqueeze(-1); // (N, H, S, D, 1)
    auto V = V_orig.unsqueeze(-2); // (N, H, S, 1, D)

    // Call the CUDA kernel
    compute_and_contract<float>(
        Q.data_ptr<float>(),
        K.data_ptr<float>(),
        V.data_ptr<float>(),
        output.data_ptr<float>(),
        N, H, S, D);
}

// PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
//     m.def("compute_and_contract", &compute_and_contract_call, "Compute and contract operation");
// }




// Debugging
#include <iostream>
#include <chrono>
// dummy main function
int main() {
    // Set the device
    torch::Device device(torch::kCUDA, 0);

    // Set the tensor dimensions
    int N = 16;
    int H = 8;
    int S = 64;
    int D = 32;

    // Create input tensors
    auto Q = torch::rand({N, H, S, D}, device);
    auto K = torch::rand({N, H, S, D}, device);
    auto V = torch::rand({N, H, S, D}, device);

    // Create output tensor
    auto output = torch::zeros({N, H, S, D}, device);

    // Call the custom CUDA kernel
    auto start = std::chrono::high_resolution_clock::now();
    compute_and_contract_call(Q, K, V, output);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Elapsed time: " << elapsed.count() << " s\n";

    return 0;
}
