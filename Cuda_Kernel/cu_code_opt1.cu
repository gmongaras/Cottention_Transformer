#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> // For hipMemcpy and hipFree
#include <torch/torch.h>
// #include <torch/extension.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <iostream>
#include <chrono>

#include <hip/hip_fp16.h> // Include CUDA half-precision definitions




// General AtomicAdd_
template<typename T>
__device__ void AtomicAdd_(T* address, T val) {
    atomicAdd(address, val);
}
// Specialization for half precision
template<>
__device__ void AtomicAdd_(at::Half* address, at::Half val) {
    atomicAdd(reinterpret_cast<__half*>(address), *reinterpret_cast<__half*>(&val));
}
// // Specialization for bfloat16 half precision
// template<>
// __device__ void AtomicAdd_(at::BFloat16* address, at::BFloat16 val) {
//     atomicAdd(reinterpret_cast<__hip_bfloat16*>(address), *reinterpret_cast<__hip_bfloat16*>(&val));
// }




// For debugging
#include <fstream>
template<typename T>
void writeTensorToFile(const std::string& filename, const T* tensorData, const std::vector<int>& shape) {
    std::ofstream file(filename, std::ios::binary | std::ios::out);

    // Write the shape
    int dimensions = shape.size();
    file.write(reinterpret_cast<const char*>(&dimensions), sizeof(dimensions));
    for (int dim : shape) {
        file.write(reinterpret_cast<const char*>(&dim), sizeof(dim));
    }

    // Get the number of elements in the tensor
    int numElements = 1;
    for (int dim : shape) numElements *= dim;

    // Allocate host memory to copy the tensor data
    size_t numBytes = numElements * sizeof(T);
    T* hostData = new T[numElements];

    // Copy the tensor data to host
    hipMemcpy(hostData, tensorData, numBytes, hipMemcpyDeviceToHost);

    // Write the tensor data to file
    file.write(reinterpret_cast<const char*>(hostData), sizeof(T) * numElements);

    // Close the file and free the host memory
    file.close();
    free(hostData);
}







// Used to do (V.unsqueeze(-1)*K.unsqueeze(-2)) for each position in the sequence
template<typename T>
__global__ void compute_outer_products(
    const T* K, const T* V, T* VK,
    int N, int H, int S, int d_V, int d_K, int s, int block_size, int BS) {
    int n = floor((float)blockIdx.x / (float)H); // Batch index
    int h = blockIdx.x % H; // Head index
    int blk_idx = blockIdx.y; // Dimension index within the sequence
    int d_v = blockIdx.z; // Dimension index within d_V
    int d_k = threadIdx.x; // Dimension index within d_k

    // Ensure we are within bounds for the d_V dimension and d_K dimension
    if (d_v < d_V && d_k < d_K) {
        // Compute indices for V and K at the current block in the sequence
        int indexV = ((n * H + h) * S + s + blk_idx) * d_V + d_v;
        int indexK = ((n * H + h) * S + s + blk_idx) * d_K + d_k;

        // Do the outer product between V and K
        T product = V[indexV] * K[indexK];

        // Iterate over all blocks with a block index greater than this one
        // and add the product to the VK tensor, thus doing a cumulative sum
        // in the shared memory.
        for (int i = blk_idx; i < BS; i++) {
            int indexVK = ((((n * H + h) * block_size + i) * d_V) + d_v) * d_K + d_k;

            // Add the product to the VK tensor
            AtomicAdd_(&VK[indexVK], product);
        }
    }
}




template<typename T>
__global__ void matrix_multiply_kernel(
    const T* Q, T* VK, T* output,
    int N, int H, int S, int d_V, int d_K, int s, int block_size, int BS) {
    int n = floor((float)blockIdx.x / (float)H); // Batch index
    int h = blockIdx.x % H; // Head index
    int blk_idx = blockIdx.y; // Dimension index within the sequence
    int d_v = blockIdx.z; // Dimension index within d_V
    int d_k = threadIdx.x; // Dimension index within d_k


    // Allocate shared memory for the cumulative sum
    extern __shared__ T shared_memory[];


    // Ensure we are within bounds for the d_V dimension and d_K dimension
    if (d_v < d_V && d_k < d_K) {
        // Compute indices for Q and VK. Note that VK does not vary with s,
        // so we use a fixed sequence index (effectively 0) for VK.
        int indexQ = ((n * H + h) * S + s + blk_idx) * d_K + d_k;

        // For VK, since it's (N, H, 1, d_V, d_K), we don't include 's' in its index calculation
        int indexVK = (((n * H + h) * block_size + blk_idx) * d_V + d_v) * d_K + d_k;


        // Multiply the Q and VK tensors and accumulate the sum in shared memory
        shared_memory[d_k] = Q[indexQ] * VK[indexVK];

        // Wait for all threads to finish writing to shared memory
        __syncthreads();

        // Only one thread sums all the elements in shared memory and stores the result in output
        if (d_k == 0) {
            int indexOutput = ((n * H + h) * S + s + blk_idx) * d_V + d_v;

            T sum_ = 0;
            for (int i = 0; i < d_K; i++) {
                sum_ += shared_memory[i];
            }
            
            atomicAdd(&output[indexOutput], sum_);
        }

        // Since each position in VK is only access once, we can copy
        // the contents of the last block to this one.
        // This ensures the cumulative sum is correct for the next block.
        // Only do this copy if the current block is not the last one
        if (blk_idx < BS-1) {
            VK[indexVK] = VK[(((n * H + h) * block_size + BS-1) * d_V + d_v) * d_K + d_k];
        }
    }
}




template<typename T>
void compute_attention(
    const T* Q, const T* K, const T* V,
    T* output,
    T* VK,
    int N, int H, int S, int d_V, int d_K,
    const int block_size,
    hipStream_t stream = 0) {
    // Grid for the matrix multiplication kernel
    // One block per batch-dimension index, head-dimension index, and both dimensions of VK

    // writeTensorToFile("Q.bin", Q, {N, H, S, d_K});
    // writeTensorToFile("K.bin", K, {N, H, S, d_K});
    // writeTensorToFile("V.bin", V, {N, H, S, d_V});

    // Iterate over the sequence dimension and compute the outer product
    for (int s = 0; s < S; s+=block_size) {
        // Block size cannot exceed the sequence length
        int BS = min(block_size, S-s);

        // Compute the cumulative product between V and K
        // up to block_size positions in the sequence
        //   Grid over N, H, and value dimension. Assuming the block size is small
        //      we can use this as the x index in the thread and y as the d_K index
        //   Threads over the number of blocks and the d_K dimension
        //   No shared memory
        //   Stream is the CUDA stream where the kernel will be executed
        dim3 grid(N*H, BS, d_V);
        compute_outer_products<T><<<grid, d_K, 0, stream>>>(K, V, VK, N, H, S, d_V, d_K, s, block_size, BS);

        // // Wait for the kernel to complete
        // hipDeviceSynchronize();

        // Product between Q at position s and VK
        //   Grid over N, H, and value dimension. Assuming the block size is small
        //      we can use this as the x index in the thread and y as the d_K index
        //   Threads over the number of blocks and the d_K dimension
        //   Shared memory is used to accumulate the sum
        //   stream - This is the CUDA stream where the kernel will be executed
        matrix_multiply_kernel<T><<<grid, d_K, d_K*sizeof(T), stream>>>(Q, VK, output, N, H, S, d_V, d_K, s, block_size, BS);

        // // Wait for the kernel to complete
        // hipDeviceSynchronize();
    }

    // writeTensorToFile("VK.bin", VK, {N, H, 1, d_V, d_K});
    // writeTensorToFile("output.bin", output, {N, H, S, d_V});
}



// Wrapper function to orchestrate the computation
template<typename T>
void compute_and_contract(
    const T* Q, const T* K, const T* V, T* output, T* VK,
    int N, int H, int S, int D,
    const int block_size,
    hipStream_t stream = 0) {
    compute_attention<T>(Q, K, V, output, VK, N, H, S, D, D, block_size, stream);
}





// // CUDA forward declarations
// void compute_and_contract_cuda(const torch::Tensor& A, const torch::Tensor& B, const torch::Tensor& C, torch::Tensor& output);
// void compute_and_contract(const torch::Tensor& A, const torch::Tensor& B, const torch::Tensor& C, torch::Tensor& output);

// C++ interface
template<typename dtype_>
torch::Tensor compute_and_contract_call(torch::Tensor& Q, torch::Tensor& K_orig, torch::Tensor& V_orig, const int block_size) {
    // Check tensor requirements, e.g., dtype, device, etc.
    TORCH_CHECK(Q.device().is_cuda(), "Q must be a CUDA tensor");
    TORCH_CHECK(K_orig.device().is_cuda(), "K must be a CUDA tensor");
    TORCH_CHECK(V_orig.device().is_cuda(), "V must be a CUDA tensor");

    // Get tensor dimensions
    int N = Q.size(0);
    int H = Q.size(1);
    int S = Q.size(2);
    int D = Q.size(3);

    // Ouput tensor
    auto output = torch::zeros({N, H, S, D}, Q.options());
    // auto output = K_orig;

    // Allocate memory for the intermediate tensors
    auto VK = torch::zeros({N, H, block_size, D, D}, Q.options());

    // writeTensorToFile("Q.bin", Q.data_ptr<float>(), {N, H, S, D});
    // writeTensorToFile("K.bin", K_orig.data_ptr<float>(), {N, H, S, D});
    // writeTensorToFile("V.bin", V_orig.data_ptr<float>(), {N, H, S, D});

    // Unsqueeze K along the last dimension and V along the second-to-last dimension
    auto K = K_orig.unsqueeze(-1); // (N, H, S, D, 1)
    auto V = V_orig.unsqueeze(-2); // (N, H, S, 1, D)

    // Ensure the tensors are contiguous
    Q = Q.contiguous();
    K = K.contiguous();
    V = V.contiguous();

    // Call the CUDA kernel
    compute_and_contract<dtype_>(
        Q.data_ptr<dtype_>(),
        K.data_ptr<dtype_>(),
        V.data_ptr<dtype_>(),
        output.data_ptr<dtype_>(),
        VK.data_ptr<dtype_>(),
        N, H, S, D, block_size);

    // writeTensorToFile("output.bin", output.data_ptr<float>(), {N, H, S, D});

    return output;
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("float32", &compute_and_contract_call<float>);
    // m.def("float16", &compute_and_contract_call<at::Half>);
    // m.def("bfloat16", &compute_and_contract_call<at::BFloat16>);
}



// // Debugging
// #include <iostream>
// #include <chrono>
// // dummy main function
// int main() {
//     // Set the device
//     torch::Device device(torch::kCUDA, 0);

//     // Set the tensor dimensions
//     int N = 16;
//     int H = 8;
//     int S = 64;
//     int D = 32;

//     // Create input tensors
//     auto Q = torch::rand({N, H, S, D}, device);
//     auto K = torch::rand({N, H, S, D}, device);
//     auto V = torch::rand({N, H, S, D}, device);

//     // Create output tensor
//     auto output = torch::zeros({N, H, S, D}, device);

//     // Call the custom CUDA kernel
//     auto start = std::chrono::high_resolution_clock::now();
//     compute_and_contract_call(Q, K, V, output, 5);
//     auto end = std::chrono::high_resolution_clock::now();
//     std::chrono::duration<double> elapsed = end - start;
//     std::cout << "Elapsed time: " << elapsed.count() << " s\n";

//     return 0;
// }
