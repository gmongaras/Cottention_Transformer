#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> // For hipMemcpy and hipFree
#include <torch/torch.h>
// #include <torch/extension.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <iostream>
#include <chrono>

#include <hip/hip_fp16.h> // Include CUDA half-precision definitions




// General AtomicAdd_
template<typename T>
__device__ void AtomicAdd_(T* address, T val) {
    atomicAdd(address, val);
}
// Specialization for half precision
template<>
__device__ void AtomicAdd_(at::Half* address, at::Half val) {
    atomicAdd(reinterpret_cast<__half*>(address), *reinterpret_cast<__half*>(&val));
}
// Specialization for bfloat16 half precision
template<>
__device__ void AtomicAdd_(at::BFloat16* address, at::BFloat16 val) {
    atomicAdd(reinterpret_cast<__hip_bfloat16*>(address), *reinterpret_cast<__hip_bfloat16*>(&val));
}




// For debugging
#include <fstream>
template<typename T>
void writeTensorToFile(const std::string& filename, const T* tensorData, const std::vector<int>& shape) {
    std::ofstream file(filename, std::ios::binary | std::ios::out);

    // Write the shape
    int dimensions = shape.size();
    file.write(reinterpret_cast<const char*>(&dimensions), sizeof(dimensions));
    for (int dim : shape) {
        file.write(reinterpret_cast<const char*>(&dim), sizeof(dim));
    }

    // Get the number of elements in the tensor
    int numElements = 1;
    for (int dim : shape) numElements *= dim;

    // Allocate host memory to copy the tensor data
    size_t numBytes = numElements * sizeof(T);
    T* hostData = new T[numElements];

    // Copy the tensor data to host
    hipMemcpy(hostData, tensorData, numBytes, hipMemcpyDeviceToHost);

    // Write the tensor data to file
    file.write(reinterpret_cast<const char*>(hostData), sizeof(T) * numElements);

    // Close the file and free the host memory
    file.close();
    free(hostData);
}







// Used to do (V.unsqueeze(-1)*K.unsqueeze(-2)) for each position in the sequence
template<typename T>
__global__ void compute_outer_products(
    const T* K, const T* V, T* VK,
    int N, int H, int S, int d_V, int d_K, int s, int block_size, int BS) {
    int n = blockIdx.x; // Batch index
    int h = blockIdx.y; // Head index
    int d_v = blockIdx.z; // Dimension index within d_V
    int blk_idx = threadIdx.x; // Dimension index within the sequence
    int d_k = threadIdx.y; // Dimension index within d_k


    // Ensure we are within bounds for the d_V dimension and d_K dimension
    if (d_v < d_V && d_k < d_K) {
        // Compute indices for V and K at the current block in the sequence
        int indexV = ((n * H + h) * S + s + blk_idx) * d_V + d_v;
        int indexK = ((n * H + h) * S + s + blk_idx) * d_K + d_k;

        // Do the outer product between V and K
        T product = V[indexV] * K[indexK];

        // Iterate over all blocks with a block index greater than this one
        // and add the product to the VK tensor, thus doing a cumulative sum
        // in the shared memory.
        for (int i = blk_idx; i < BS; i++) {
            int indexVK = ((((n * H + h) * block_size + i) * d_V) + d_v) * d_K + d_k;

            // Add the product to the VK tensor
            AtomicAdd_(&VK[indexVK], product);
        }
    }
}




template<typename T>
__global__ void matrix_multiply_kernel(
    const T* Q, T* VK, T* output,
    int N, int H, int S, int d_V, int d_K, int s, int block_size, int BS) {
    int n = blockIdx.x; // Batch index
    int h = blockIdx.y; // Head index
    int d_v = blockIdx.z; // Dimension index within d_V
    int blk_idx = threadIdx.x; // Sequence index
    int d_k = threadIdx.y; // Dimension index within d_K

    // Ensure we are within bounds for the d_V dimension and d_K dimension
    if (d_v < d_V && d_k < d_K) {
        // Compute indices for Q and VK. Note that VK does not vary with s,
        // so we use a fixed sequence index (effectively 0) for VK.
        int indexQ = ((n * H + h) * S + s + blk_idx) * d_K + d_k;

        // For VK, since it's (N, H, 1, d_V, d_K), we don't include 's' in its index calculation
        int indexVK = (((n * H + h) * block_size + blk_idx) * d_V + d_v) * d_K + d_k;

        // Element-wise multiplication and add to shared memory
        // Write the accumulated sum to the output tensor
        int indexOutput = ((n * H + h) * S + s + blk_idx) * d_V + d_v;
        AtomicAdd_(&output[indexOutput], Q[indexQ] * VK[indexVK]);

        // Since each position in VK is only access once, we can copy
        // the contents of the last block to this one.
        // This ensures the cumulative sum is correct for the next block.
        VK[indexVK] = VK[(((n * H + h) * block_size + BS-1) * d_V + d_v) * d_K + d_k];
    }
}




template<typename T>
void compute_attention(
    const T* Q, const T* K, const T* V,
    T* output,
    T* VK,
    int N, int H, int S, int d_V, int d_K,
    const int block_size,
    hipStream_t stream = 0) {
    // Grid for the matrix multiplication kernel
    // One block per batch-dimension index, head-dimension index, and both dimensions of VK
    dim3 grid(N, H, d_V);

    // writeTensorToFile("Q.bin", Q, {N, H, S, d_K});
    // writeTensorToFile("K.bin", K, {N, H, S, d_K});
    // writeTensorToFile("V.bin", V, {N, H, S, d_V});

    // Iterate over the sequence dimension and compute the outer product
    for (int s = 0; s < S; s+=block_size) {
        // Block size cannot exceed the sequence length
        int BS = min(block_size, S-s);

        // Compute the cumulative product between V and K
        // up to block_size positions in the sequence
        //   Grid over N, H, and value dimension. Assuming the block size is small
        //      we can use this as the x index in the thread and y as the d_K index
        //   Threads over the number of blocks and the d_K dimension
        //   No shared memory
        //   Stream is the CUDA stream where the kernel will be executed
        compute_outer_products<T><<<grid, {BS, d_K}, 0, stream>>>(K, V, VK, N, H, S, d_V, d_K, s, block_size, BS);

        // Wait for the kernel to complete
        hipDeviceSynchronize();

        // Product between Q at position s and VK
        //   Grid over N, H, and value dimension. Assuming the block size is small
        //      we can use this as the x index in the thread and y as the d_K index
        //   Threads over the number of blocks and the d_K dimension
        //   0 - This is the shared memory size, which is not used in this kernel
        //   stream - This is the CUDA stream where the kernel will be executed
        matrix_multiply_kernel<T><<<grid, {BS, d_K}, 0, stream>>>(Q, VK, output, N, H, S, d_V, d_K, s, block_size, BS);

        // Wait for the kernel to complete
        hipDeviceSynchronize();
    }

    // writeTensorToFile("VK.bin", VK, {N, H, 1, d_V, d_K});
    // writeTensorToFile("output.bin", output, {N, H, S, d_V});
}



// Wrapper function to orchestrate the computation
template<typename T>
void compute_and_contract(
    const T* Q, const T* K, const T* V, T* output, T* VK,
    int N, int H, int S, int D,
    const int block_size,
    hipStream_t stream = 0) {
    compute_attention<T>(Q, K, V, output, VK, N, H, S, D, D, block_size, stream);
}





// // CUDA forward declarations
// void compute_and_contract_cuda(const torch::Tensor& A, const torch::Tensor& B, const torch::Tensor& C, torch::Tensor& output);
// void compute_and_contract(const torch::Tensor& A, const torch::Tensor& B, const torch::Tensor& C, torch::Tensor& output);

// C++ interface
template<typename dtype_>
torch::Tensor compute_and_contract_call(const torch::Tensor& Q, const torch::Tensor& K_orig, const torch::Tensor& V_orig, const int block_size) {
    // Check tensor requirements, e.g., dtype, device, etc.
    TORCH_CHECK(Q.device().is_cuda(), "Q must be a CUDA tensor");
    TORCH_CHECK(K_orig.device().is_cuda(), "K must be a CUDA tensor");
    TORCH_CHECK(V_orig.device().is_cuda(), "V must be a CUDA tensor");

    // Get tensor dimensions
    int N = Q.size(0);
    int H = Q.size(1);
    int S = Q.size(2);
    int D = Q.size(3);

    // Ouput tensor
    auto output = torch::zeros({N, H, S, D}, Q.options());

    // Allocate memory for the intermediate tensors
    auto VK = torch::zeros({N, H, block_size, D, D}, Q.options());

    // writeTensorToFile("Q.bin", Q.data_ptr<float>(), {N, H, S, D});
    // writeTensorToFile("K.bin", K_orig.data_ptr<float>(), {N, H, S, D});
    // writeTensorToFile("V.bin", V_orig.data_ptr<float>(), {N, H, S, D});

    // Unsqueeze K along the last dimension and V along the second-to-last dimension
    auto K = K_orig.unsqueeze(-1); // (N, H, S, D, 1)
    auto V = V_orig.unsqueeze(-2); // (N, H, S, 1, D)

    // Call the CUDA kernel
    compute_and_contract<dtype_>(
        Q.data_ptr<dtype_>(),
        K.data_ptr<dtype_>(),
        V.data_ptr<dtype_>(),
        output.data_ptr<dtype_>(),
        VK.data_ptr<dtype_>(),
        N, H, S, D, block_size);

    // writeTensorToFile("output.bin", output.data_ptr<float>(), {N, H, S, D});

    return output;
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("float32", &compute_and_contract_call<float>);
    m.def("float16", &compute_and_contract_call<at::Half>);
    m.def("bfloat16", &compute_and_contract_call<at::BFloat16>);
}



// // Debugging
// #include <iostream>
// #include <chrono>
// // dummy main function
// int main() {
//     // Set the device
//     torch::Device device(torch::kCUDA, 0);

//     // Set the tensor dimensions
//     int N = 16;
//     int H = 8;
//     int S = 64;
//     int D = 32;

//     // Create input tensors
//     auto Q = torch::rand({N, H, S, D}, device);
//     auto K = torch::rand({N, H, S, D}, device);
//     auto V = torch::rand({N, H, S, D}, device);

//     // Create output tensor
//     auto output = torch::zeros({N, H, S, D}, device);

//     // Call the custom CUDA kernel
//     auto start = std::chrono::high_resolution_clock::now();
//     compute_and_contract_call(Q, K, V, output, 5);
//     auto end = std::chrono::high_resolution_clock::now();
//     std::chrono::duration<double> elapsed = end - start;
//     std::cout << "Elapsed time: " << elapsed.count() << " s\n";

//     return 0;
// }
