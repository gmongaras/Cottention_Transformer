#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> // For hipMemcpy and hipFree
#include <torch/torch.h>
#include <torch/extension.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <iostream>
#include <chrono>






// For debugging
#include <fstream>
template<typename T>
void writeTensorToFile(const std::string& filename, const T* tensorData, const std::vector<int>& shape) {
    std::ofstream file(filename, std::ios::binary | std::ios::out);

    // Write the shape
    int dimensions = shape.size();
    file.write(reinterpret_cast<const char*>(&dimensions), sizeof(dimensions));
    for (int dim : shape) {
        file.write(reinterpret_cast<const char*>(&dim), sizeof(dim));
    }

    // Get the number of elements in the tensor
    int numElements = 1;
    for (int dim : shape) numElements *= dim;

    // Allocate host memory to copy the tensor data
    size_t numBytes = numElements * sizeof(T);
    T* hostData = new T[numElements];

    // Copy the tensor data to host
    hipMemcpy(hostData, tensorData, numBytes, hipMemcpyDeviceToHost);

    // Write the tensor data to file
    file.write(reinterpret_cast<const char*>(hostData), sizeof(T) * numElements);

    // Close the file and free the host memory
    file.close();
    free(hostData);
}






// Used to do (V.unsqueeze(-1)*K.unsqueeze(-2))
template<typename T>
__global__ void compute_outer_product_kernel(
    const T* K, const T* V, T* VK,
    int N, int H, int S, int d_V, int d_K, int s) {
    int n = blockIdx.x; // Batch index
    int h = blockIdx.y; // Head index

    // Each thread handles a portion of the elements in VK
    int threadId = threadIdx.x;
    int totalThreads = blockDim.x;

    for (int idx = threadId; idx < d_V * d_K; idx += totalThreads) {
        int d_v = idx % d_V; // Correct calculation for Dimension index for V
        int d_k = idx / d_V; // Correct calculation for Dimension index for K

        // Compute indices for V, K, and VK
        int indexV = ((n * H + h) * S + s) * d_V + d_v;
        int indexK = ((n * H + h) * S + s) * d_K + d_k;
        int indexVK = ((((n * H + h) * 1 + 0) * d_V) + d_v) * d_K + d_k; // Use s=0 for VK since it's the accumulation target

        // Perform the outer product and store in VK directly without addition
        VK[indexVK] += V[indexV] * K[indexK];
    }
}




template<typename T>
__global__ void matrix_multiply_kernel(
    const T* Q, const T* VK, T* output,
    int N, int H, int S, int d_V, int d_K, int s) {
    int n = blockIdx.x; // Batch index
    int h = blockIdx.y; // Head index
    int d_v = threadIdx.x; // Dimension index within d_V (output dimension)

    // Ensure we are within bounds for the d_V dimension
    if (d_v < d_V) {
        T sum = 0;
        // Perform the dot product along the D dimension (which corresponds to d_K in VK)
        for (int D_idx = 0; D_idx < d_K; ++D_idx) {
            // Compute indices for Q and VK. Note that VK does not vary with s,
            // so we use a fixed sequence index (effectively 0) for VK.
            int indexQ = ((n * H + h) * S + s) * d_K + D_idx;
            // For VK, since it's (N, H, 1, d_V, d_K), we don't include 's' in its index calculation
            int indexVK = (((n * H + h) * 1 + 0) * d_V + d_v) * d_K + D_idx;

            // Perform element-wise multiplication and accumulate the sum
            sum += Q[indexQ] * VK[indexVK];
        }

        // Write the accumulated sum to the output tensor
        int indexOutput = ((n * H + h) * S + s) * d_V + d_v;
        output[indexOutput] = sum;
    }
}




template<typename T>
void compute_attention(
    const T* Q, const T* K, const T* V,
    T* output,
    int N, int H, int S, int d_V, int d_K,
    hipStream_t stream = 0) {
    // Fixed number of threads per block
    int threadsPerBlock = 1024;

    // One block per batch-dimension index and head-dimension index
    dim3 grid(N, H);

    // Intermediate tensor to store the product between V and K
    // at each position in the sequence
    // The shape is (N, H, 1, d_V, d_K)
    T* VK;
    // Initialize to zeros
    hipMalloc(&VK, N * H * 1 * d_V * d_K * sizeof(T));
    hipMemset(VK, 0, N * H * 1 * d_V * d_K * sizeof(T));

    // writeTensorToFile("Q.bin", Q, {N, H, S, d_K});
    // writeTensorToFile("K.bin", K, {N, H, S, d_K});
    // writeTensorToFile("V.bin", V, {N, H, S, d_V});

    // Iterate over the sequence dimension and compute the outer product
    for (int s = 0; s < S; ++s) {
        // Launch the kernel
        // This will compute the outer product between V and K at each position in the sequence
        // and add the result to VK
        compute_outer_product_kernel<T><<<grid, threadsPerBlock, 0, stream>>>(K, V, VK, N, H, S, d_V, d_K, s);

        // Wait for the kernel to complete
        hipDeviceSynchronize();

        // Product between Q at position s and VK
        // This is the output for the s-th position in the sequence
        matrix_multiply_kernel<T><<<grid, d_V, 0, stream>>>(Q, VK, output, N, H, S, d_V, d_K, s);

        // Wait for the kernel to complete
        hipDeviceSynchronize();
    }

    // writeTensorToFile("VK.bin", VK, {N, H, 1, d_V, d_K});
    // writeTensorToFile("output.bin", output, {N, H, S, d_V});

    // Free the intermediate tensor
    hipFree(VK);
}



// Wrapper function to orchestrate the computation
template<typename T>
void compute_and_contract(
    const T* Q, const T* K, const T* V, T* output,
    int N, int H, int S, int D,
    hipStream_t stream = 0) {
    int threadsPerBlock = 256; // Example, adjust based on device capabilities
    int blocksPerGrid = (N * H * S * D + threadsPerBlock - 1) / threadsPerBlock;

    compute_attention<T>(Q, K, V, output, N, H, S, D, D, stream);
}





// // CUDA forward declarations
// void compute_and_contract_cuda(const torch::Tensor& A, const torch::Tensor& B, const torch::Tensor& C, torch::Tensor& output);
// void compute_and_contract(const torch::Tensor& A, const torch::Tensor& B, const torch::Tensor& C, torch::Tensor& output);

// C++ interface
void compute_and_contract_call(const torch::Tensor& Q, const torch::Tensor& K_orig, const torch::Tensor& V_orig, torch::Tensor& output) {
    // Check tensor requirements, e.g., dtype, device, etc.
    TORCH_CHECK(Q.device().is_cuda(), "Q must be a CUDA tensor");
    TORCH_CHECK(K_orig.device().is_cuda(), "K must be a CUDA tensor");
    TORCH_CHECK(V_orig.device().is_cuda(), "V must be a CUDA tensor");
    TORCH_CHECK(output.device().is_cuda(), "output must be a CUDA tensor");

    // Get tensor dimensions
    int N = Q.size(0);
    int H = Q.size(1);
    int S = Q.size(2);
    int D = Q.size(3);

    // writeTensorToFile("Q.bin", Q.data_ptr<float>(), {N, H, S, D});
    // writeTensorToFile("K.bin", K_orig.data_ptr<float>(), {N, H, S, D});
    // writeTensorToFile("V.bin", V_orig.data_ptr<float>(), {N, H, S, D});

    // Unsqueeze K along the last dimension and V along the second-to-last dimension
    auto K = K_orig.unsqueeze(-1); // (N, H, S, D, 1)
    auto V = V_orig.unsqueeze(-2); // (N, H, S, 1, D)

    // Call the CUDA kernel
    compute_and_contract<float>(
        Q.data_ptr<float>(),
        K.data_ptr<float>(),
        V.data_ptr<float>(),
        output.data_ptr<float>(),
        N, H, S, D);

    // writeTensorToFile("output.bin", output.data_ptr<float>(), {N, H, S, D});
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("compute_and_contract", &compute_and_contract_call, "Compute and contract operation");
}




// Debugging
#include <iostream>
#include <chrono>
// dummy main function
int main() {
    // Set the device
    torch::Device device(torch::kCUDA, 0);

    // Set the tensor dimensions
    int N = 16;
    int H = 8;
    int S = 64;
    int D = 32;

    // Create input tensors
    auto Q = torch::rand({N, H, S, D}, device);
    auto K = torch::rand({N, H, S, D}, device);
    auto V = torch::rand({N, H, S, D}, device);

    // Create output tensor
    auto output = torch::zeros({N, H, S, D}, device);

    // Call the custom CUDA kernel
    auto start = std::chrono::high_resolution_clock::now();
    compute_and_contract_call(Q, K, V, output);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Elapsed time: " << elapsed.count() << " s\n";

    return 0;
}
