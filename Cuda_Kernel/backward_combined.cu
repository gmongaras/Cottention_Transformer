#include "hip/hip_runtime.h"
// Hehe
// ^w^
// UwU
// OwO
// Nyaa~ <- What I say when I'm coding in cuda. Nya~~~
// Rawr~
// >w<
// >_<
// >.<
// >:3 <- Doesn't matter if it's a cat or a dog. It's a catdog.
// >:D 
// >:P
// >:( <- This me. IDK how to code cuda kernels.
// ^w^
// UwU
// Nya~~


// ⠀⢸⠂⠀⠀⠀⠘⣧⠀⠀⣟⠛⠲⢤⡀⠀⠀⣰⠏⠀⠀⠀⠀⠀⢹⡀
// ⠀⡿⠀⠀⠀⠀⠀⠈⢷⡀⢻⡀⠀⠀⠙⢦⣰⠏⠀⠀⠀⠀⠀⠀⢸⠀
// ⠀⡇⠀⠀⠀⠀⠀⠀⢀⣻⠞⠛⠀⠀⠀⠀⠻⠀⠀⠀⠀⠀⠀⠀⢸⠀
// ⠀⡇⠀⠀⠀⠀⠀⠀⠛⠓⠒⠓⠓⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⢸⠀
// ⠀⡇⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⣸⠀
// ⠀⢿⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⢀⣀⣀⣀⣀⠀⠀⢀⡟⠀
// ⠀⠘⣇⠀⠘⣿⠋⢹⠛⣿⡇⠀⠀⠀⠀⣿⣿⡇⠀⢳⠉⠀⣠⡾⠁⠀
// ⣦⣤⣽⣆⢀⡇⠀⢸⡇⣾⡇⠀⠀⠀⠀⣿⣿⡷⠀⢸⡇⠐⠛⠛⣿⠀
// ⠹⣦⠀⠀⠸⡇⠀⠸⣿⡿⠁⢀⡀⠀⠀⠿⠿⠃⠀⢸⠇⠀⢀⡾⠁⠀
// ⠀⠈⡿⢠⢶⣡⡄⠀⠀⠀⠀⠉⠁⠀⠀⠀⠀⠀⣴⣧⠆⠀⢻⡄⠀⠀
// ⠀⢸⠃⠀⠘⠉⠀⠀⠀⠠⣄⡴⠲⠶⠴⠃⠀⠀⠀⠉⡀⠀⠀⢻⡄⠀
// ⠀⠘⠒⠒⠻⢦⣄⡀⠀⠀⠀⠀⠀⠀⠀⠀⢀⣀⣤⠞⠛⠒⠛⠋⠁⠀
// ⠀⠀⠀⠀⠀⠀⠸⣟⠓⠒⠂⠀⠀⠀⠀⠀⠈⢷⡀⠀⠀⠀⠀⠀⠀⠀
// ⠀⠀⠀⠀⠀⠀⠀⠙⣦⠀⠀⠀⠀⠀⠀⠀⠀⠈⢷⠀⠀⠀⠀⠀⠀⠀
// ⠀⠀⠀⠀⠀⠀⠀⣼⣃⡀⠀⠀⠀⠀⠀⠀⠀⠀⠘⣆⠀⠀⠀⠀⠀⠀
// ⠀⠀⠀⠀⠀⠀⠀⠉⣹⠃⠀⠀⠀⠀⠀⠀⠀⠀⠀⢻⠀⠀⠀⠀⠀⠀
// ⠀⠀⠀⠀⠀⠀⠀⠀⡿⠀⠀⠀⠀⠀⠀⠀⠀⠀⠀⢸⡆⠀⠀⠀⠀⠀
// OOOH you like coding cuda kernels? You're an insane person. UwU




#include <hip/hip_runtime.h> // For hipMemcpy and hipFree
#include <torch/torch.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/autocast_mode.h>
// #include <torch/extension.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <iostream>
#include <chrono>

#include <hip/hip_fp16.h> // Include CUDA half-precision definitions




// General AtomicAdd_
template<typename T>
__device__ void AtomicAdd_(T* address, T val) {
    atomicAdd(address, val);
}
// Specialization for half precision
template<>
__device__ void AtomicAdd_(at::Half* address, at::Half val) {
    atomicAdd(reinterpret_cast<__half*>(address), *reinterpret_cast<__half*>(&val));
}
// Specialization for bfloat16 half precision
template<>
__device__ void AtomicAdd_(at::BFloat16* address, at::BFloat16 val) {
    atomicAdd(reinterpret_cast<__hip_bfloat16*>(address), *reinterpret_cast<__hip_bfloat16*>(&val));
}












// General __shfl_down_sync
template<typename T>
__device__ T __shfl_down_sync_(unsigned mask, T val, int delta, int width = warpSize) {
    return __shfl_down_sync(mask, val, delta, width);
}
// Specialization for half precision
template<>
__device__ at::Half __shfl_down_sync_(unsigned mask, at::Half val, int delta, int width) {
    return __shfl_down_sync(mask, *reinterpret_cast<__half*>(&val), delta, width);
}
// Specialization for bfloat16 half precision
template<>
__device__ at::BFloat16 __shfl_down_sync_(unsigned mask, at::BFloat16 val, int delta, int width) {
    return __shfl_down_sync(mask, *reinterpret_cast<__hip_bfloat16*>(&val), delta, width);
}






































template <typename T>
__inline__ __device__ void warpReduceSum(T& val1, T& val2) {
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        val1 += __shfl_down_sync_(0xffffffff, val1, offset);
        val2 += __shfl_down_sync_(0xffffffff, val2, offset);
    }
}

template <typename T>
__inline__ __device__ void blockReduce(T& val1, T& val2) {
    static __shared__ T shared1[32]; // For the first value
    static __shared__ T shared2[32]; // For the second value
    int lane = threadIdx.x % warpSize;
    int wid = threadIdx.x / warpSize;

    // Reduce within each warp for both values
    warpReduceSum(val1, val2);

    // Write reduced values to shared memory
    if (lane == 0) {
        shared1[wid] = val1;
        shared2[wid] = val2;
    }

    __syncthreads();

    // Use only the first warp for the final reduction
    val1 = (threadIdx.x < blockDim.x / warpSize) ? shared1[lane] : (T)0;
    val2 = (threadIdx.x < blockDim.x / warpSize) ? shared2[lane] : (T)0;
    if (wid == 0) {
        warpReduceSum(val1, val2); // Final reduce within the first warp
    }

    __syncthreads();
}






template<typename T>
__inline__ __device__ void backward_kernel_double_over_d__v_cache_one_call(
    const T* Q, const T* K, const T* V, const T* prev_grad,
    T* K_grad, T* V_grad,
    T* shared_memory_cumsum_GQ, T* shared_memory_reduce_Vgrad,
    T* shared_memory_cumsum_QG, T* shared_memory_reduce_Kgrad,
    T* shared_memory_GCache, T* shared_memory_QCache,
    int s, int S, int n, int N, int h, int H, int d_g, int d_G, int d_q, int d_Q ) {
    

    // Wait for all threads to finish the previous iteration
    __syncthreads();

    // 1: Each thread computes G[:, :, s, d_g] * Q[:, :, s, d_q] and adds it to shared[d_q] (block 1)
    //    and each thread computes Q[:, :, s, d_g] * G[:, :, s, d_v] and adds it to shared[2*d_Q + d_q] (block 3)
    shared_memory_cumsum_GQ[d_q] += prev_grad[((n * H + h) * S + s) * d_G + d_g] * Q[((n * H + h) * S + s) * d_Q + d_q];
    shared_memory_cumsum_QG[d_q] += Q[((n * H + h) * S + s) * d_G + d_g] * prev_grad[((n * H + h) * S + s) * d_Q + d_q];

    // 2: Multiply shared[d_q] by K[:, :, s, d_q] and store it in the second part of the shared memory shared[d_Q + d_q] for the V gradient
    //    and multiply shared[2*d_Q + d_q] by V[:, :, s, d_q] and store it in the third part of the shared memory shared[3*d_Q + d_q] for the K gradient
    shared_memory_reduce_Vgrad[d_q] = shared_memory_cumsum_GQ[d_q] * K[((n * H + h) * S + s) * d_Q + d_q];
    shared_memory_reduce_Kgrad[d_q] = shared_memory_cumsum_QG[d_q] * V[((n * H + h) * S + s) * d_Q + d_q];

    // 3: Thread 1 sums all the elements in block 1 and store it in grad_V[:, :, s, d_g]
    //    Thread 2 sums all the elements in block 4 and store it in grad_K[:, :, s, d_g]
    __syncthreads();
    // if (d_q == 0) {
    //     T sum_ = 0;
    //     for (int i = 0; i < d_Q; i++) {
    //         sum_ += shared_memory_reduce_Vgrad[i];
    //     }
    //     V_grad[((n * H + h) * S + s) * d_G + d_g] = sum_;
    // }
    // else if (d_q == 1) {
    //     T sum_ = 0;
    //     for (int i = 0; i < d_Q; i++) {
    //         sum_ += shared_memory_reduce_Kgrad[i];
    //     }
    //     K_grad[((n * H + h) * S + s) * d_G + d_g] = sum_;
    // }

    // Array of two values to store the output of the blockReduce
    blockReduce(shared_memory_reduce_Vgrad[d_q], shared_memory_reduce_Kgrad[d_q]);
    if (threadIdx.x == 0) {
        V_grad[((n * H + h) * S + s) * d_G + d_g] = shared_memory_reduce_Vgrad[d_q];
        K_grad[((n * H + h) * S + s) * d_G + d_g] = shared_memory_reduce_Kgrad[d_q];
    }
}



template<typename T, unsigned int inner_loop_size>
__inline__ __device__ void backward_kernel_double_over_d__v_cache_loop_inner(
    const T* Q, const T* K, const T* V, const T* prev_grad,
    T* K_grad, T* V_grad,
    T* shared_memory_cumsum_GQ, T* shared_memory_reduce_Vgrad,
    T* shared_memory_cumsum_QG, T* shared_memory_reduce_Kgrad,
    T* shared_memory_GCache, T* shared_memory_QCache,
    int s_start, int S, int n, int N, int h, int H, int d_g, int d_G, int d_q, int d_Q ) {
    #pragma unroll
    for (int i = 0; i < inner_loop_size; i++) {
        backward_kernel_double_over_d__v_cache_one_call<T>(
            Q, K, V, prev_grad, K_grad, V_grad, shared_memory_cumsum_GQ, shared_memory_reduce_Vgrad, shared_memory_cumsum_QG, shared_memory_reduce_Kgrad, shared_memory_GCache, shared_memory_QCache, s_start-i, S, n, N, h, H, d_g, d_G, d_q, d_Q);
    }
    // forward_kernel_double_over_d__v_cache_one_call<T>(
    //         Q, K, V, output, shared_memory_cumsum, shared_memory_reduce, shared_memory_VCache, s_start, S, n, N, h, H, d_k, d_K, d_v, d_V);
    // forward_kernel_double_over_d__v_cache_one_call<T>(
    //         Q, K, V, output, shared_memory_cumsum, shared_memory_reduce, shared_memory_VCache, s_start+1, S, n, N, h, H, d_k, d_K, d_v, d_V);
    // forward_kernel_double_over_d__v_cache_one_call<T>(
    //         Q, K, V, output, shared_memory_cumsum, shared_memory_reduce, shared_memory_VCache, s_start+2, S, n, N, h, H, d_k, d_K, d_v, d_V);
    // forward_kernel_double_over_d__v_cache_one_call<T>(
    //         Q, K, V, output, shared_memory_cumsum, shared_memory_reduce, shared_memory_VCache, s_start+3, S, n, N, h, H, d_k, d_K, d_v, d_V);
    // forward_kernel_double_over_d__v_cache_one_call<T>(
    //         Q, K, V, output, shared_memory_cumsum, shared_memory_reduce, shared_memory_VCache, s_start+4, S, n, N, h, H, d_k, d_K, d_v, d_V);
    // forward_kernel_double_over_d__v_cache_one_call<T>(
    //         Q, K, V, output, shared_memory_cumsum, shared_memory_reduce, shared_memory_VCache, s_start+5, S, n, N, h, H, d_k, d_K, d_v, d_V);
    // forward_kernel_double_over_d__v_cache_one_call<T>(
    //         Q, K, V, output, shared_memory_cumsum, shared_memory_reduce, shared_memory_VCache, s_start+6, S, n, N, h, H, d_k, d_K, d_v, d_V);
    // forward_kernel_double_over_d__v_cache_one_call<T>(
    //         Q, K, V, output, shared_memory_cumsum, shared_memory_reduce, shared_memory_VCache, s_start+7, S, n, N, h, H, d_k, d_K, d_v, d_V);
}





template<typename T, unsigned int inner_loop_size>
__global__ void backward_kernel_double_over_d__v_cache(
    const T* Q, const T* K, const T* V, const T* prev_grad,
    T* K_grad, T* V_grad,
    int N, int H, int S, int d_G, int d_Q,
    const int block_size) {
    
    int n = blockIdx.y; // Batch index
    int h = blockIdx.z; // Head index
    int d_g = blockIdx.x; // Dimension index within d_G
    int d_q = threadIdx.x; // Dimension index within d_Q

    int shared_memory_row_size = 2 * d_Q;


    // // Ensure we are within bounds
    // if (d_k >= d_K || d_v >= d_V) {
    //     return;
    // }


    // Allocate shared memory
    // My man!
    // https://github.com/pytorch/extension-cpp/issues/59#issuecomment-626189915
    // extern __shared__ __align__(sizeof(T)) unsigned char shared_memory_uchar[];
    // T *shared_memory = reinterpret_cast<T *>(shared_memory_uchar);
    extern __shared__ __align__(sizeof(T)) unsigned char shared_memory_uchar[];T *shared_memory_cumsum_GQ = reinterpret_cast<T *>(shared_memory_uchar);
    T* shared_memory_reduce_Vgrad = &shared_memory_cumsum_GQ[d_Q];
    T* shared_memory_cumsum_QG = &shared_memory_cumsum_GQ[2*d_Q];
    T* shared_memory_reduce_Kgrad = &shared_memory_cumsum_GQ[3*d_Q];
    T* shared_memory_GCache = &shared_memory_cumsum_GQ[4*d_Q];
    T* shared_memory_QCache = &shared_memory_cumsum_GQ[4*d_Q + S];

    // Initialize the shared memory to 0
    if (d_q < d_Q) {
        shared_memory_cumsum_GQ[d_q] = shared_memory_cumsum_QG[d_q] =  0;
    }



    // Cache the previous gradient and Q values
    for (int s = d_q; s < S; s += d_Q) {
        shared_memory_GCache[s] = prev_grad[((n * H + h) * S + s) * d_G + d_g];
        shared_memory_QCache[s] = Q[((n * H + h) * S + s) * d_Q + d_q];
    }
    __syncthreads();


    // Iterate over the first uneven part of the sequence
    for (int s = S-1; s >= floor((float)S/(float)inner_loop_size)*inner_loop_size; s--) {
        backward_kernel_double_over_d__v_cache_one_call<T>(
            Q, K, V, prev_grad, K_grad, V_grad, shared_memory_cumsum_GQ, shared_memory_reduce_Vgrad, shared_memory_cumsum_QG, shared_memory_reduce_Kgrad, shared_memory_GCache, shared_memory_QCache, s, S, n, N, h, H, d_g, d_G, d_q, d_Q);
    }

    // Iterate over the entire sequence
    for (int s = floor((float)S/(float)inner_loop_size); s > 0; s--) {
        backward_kernel_double_over_d__v_cache_loop_inner<T, inner_loop_size>(
            Q, K, V, prev_grad, K_grad, V_grad, shared_memory_cumsum_GQ, shared_memory_reduce_Vgrad, shared_memory_cumsum_QG, shared_memory_reduce_Kgrad, shared_memory_GCache, shared_memory_QCache, s*inner_loop_size-1, S, n, N, h, H, d_g, d_G, d_q, d_Q);
    }
}



// Wrapper function to orchestrate the computation
template<typename T>
void backward_call_double_over_d__v_cache(
    const T* Q, const T* K, const T* V, const T* prev_grad,
    T* K_grad, T* V_grad,
    int N, int H, int S, int D,
    const int block_size,
    hipStream_t stream = 0) {

    int d_V = D;
    int d_K = D;

    // Inner loop size is 8
    const int inner_loop_size = 8;

    // Shared memory has 4 parts:
    //  1: Cumulative sum of G_j * Q over all Q
    //  2: block 1 multiplied by K for the V gradient
    //  3: Cumulative sum of Q_j * G over all G
    //  4: block 3 multiplied by V for the K gradient
    //  5: Cache for the prev_grad
    //  6: Cache for the Q values
    dim3 grid(d_V, N, H);
    dim3 block((int)d_K);
    backward_kernel_double_over_d__v_cache<T, inner_loop_size><<<grid, block, 4*d_K*sizeof(T) + 2*S*sizeof(T), stream>>>(Q, K, V, prev_grad, K_grad, V_grad, N, H, S, D, D, block_size);
}


















































// C++ interface
template<typename dtype_>
std::vector<torch::Tensor> backward_(torch::Tensor& Q, torch::Tensor& K, torch::Tensor& V, torch::Tensor& prev_grad, const int8_t block_size) {
    // Check tensor requirements, e.g., dtype, device, etc.
    TORCH_CHECK(Q.device().is_cuda(), "Q must be a CUDA tensor");
    TORCH_CHECK(K.device().is_cuda(), "K must be a CUDA tensor");
    TORCH_CHECK(V.device().is_cuda(), "V must be a CUDA tensor");
    TORCH_CHECK(prev_grad.device().is_cuda(), "prev_grad must be a CUDA tensor");

    // Get tensor dimensions
    int N = Q.size(0);
    int H = Q.size(1);
    int S = Q.size(2);
    int D = Q.size(3);

    // Get the data type, could be auto casted
    auto data_type = at::autocast::is_enabled() && Q.scalar_type() == at::kFloat ? at::kHalf : Q.scalar_type();

    // // Unsqueeze prev_grad along the last dimension and Q along the second-to-last dimension
    // auto prev_grad = prev_grad_orig.unsqueeze(-1); // (N, H, S, D, 1)
    // auto V = V_orig.unsqueeze(-2); // (N, H, S, 1, D)
    // Unsqueeze not needed as I am making the kernel hehe UwU

    // Ensure the tensors are contiguous
    Q = Q.contiguous().to(data_type);
    K = K.contiguous().to(data_type);
    V = V.contiguous().to(data_type);
    prev_grad = prev_grad.contiguous().to(data_type);

    // Ouput tensors, gradient of K and V
    auto K_grad = torch::zeros({N, H, S, D}, torch::TensorOptions().dtype(data_type).device(Q.device()));
    auto V_grad = torch::zeros({N, H, S, D}, torch::TensorOptions().dtype(data_type).device(Q.device()));

    // https://github.com/Dao-AILab/flash-attention/blob/main/csrc/flash_attn/flash_api.cpp
    // Otherwise the kernel will be launched from cuda:0 device
    // Cast to char to avoid compiler warning about narrowing
    at::cuda::CUDAGuard device_guard{(char)Q.get_device()};

    // Using AT_DISPATCH_FLOATING_TYPES_AND_HALF to handle different data types
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(Q.scalar_type(), "backward_cuda", ([&] {
        backward_call_double_over_d__v_cache<scalar_t>(
        Q.data_ptr<scalar_t>(),
        K.data_ptr<scalar_t>(),
        V.data_ptr<scalar_t>(),
        prev_grad.data_ptr<scalar_t>(),
        K_grad.data_ptr<scalar_t>(),
        V_grad.data_ptr<scalar_t>(),
        N, H, S, D, block_size);
    }));

    return {K_grad, V_grad};
}




TORCH_LIBRARY_IMPL(TORCH_EXTENSION_NAME, Autocast, m) {
    m.impl("float32", backward_<float>);
    m.impl("float64", backward_<double>);
    m.impl("float16", backward_<at::Half>);
    try {
        m.impl("bfloat16", backward_<at::BFloat16>);
    } catch (const std::exception& e) {
        std::cout << "GPU does not support bfloat16. Skipping..." << std::endl;
        // std::cerr << "Error: " << e.what() << std::endl;
    }
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("float32", &backward_<float>);
    m.def("float64", &backward_<double>);
    m.def("float16", &backward_<at::Half>);
    try {
        m.def("bfloat16", &backward_<at::BFloat16>);
    } catch (const std::exception& e) {
        std::cout << "GPU does not support bfloat16. Skipping..." << std::endl;
        // std::cerr << "Error: " << e.what() << std::endl;
    }
}



// // Debugging
// #include <iostream>
// #include <chrono>
// // dummy main function
// int main() {
//     // Set the device
//     torch::Device device(torch::kCUDA, 0);

//     // Set the tensor dimensions
//     int N = 16;
//     int H = 8;
//     int S = 64;
//     int D = 32;

//     // Create input tensors
//     auto Q = torch::rand({N, H, S, D}, device);
//     auto K = torch::rand({N, H, S, D}, device);
//     auto V = torch::rand({N, H, S, D}, device);

//     // Create output tensor
//     auto output = torch::zeros({N, H, S, D}, device);

//     // Call the custom CUDA kernel
//     auto start = std::chrono::high_resolution_clock::now();
//     compute_and_contract_call(Q, K, V, output, 5);
//     auto end = std::chrono::high_resolution_clock::now();
//     std::chrono::duration<double> elapsed = end - start;
//     std::cout << "Elapsed time: " << elapsed.count() << " s\n";

//     return 0;
// }




// ⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿
// ⣿⣿⣿⣿⣿⣿⣿⡿⠿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿
// ⣿⣿⣿⣿⣿⣿⡟⠀⣠⣀⠙⠿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⠀⣄⠈⠻⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿
// ⣿⣿⣿⣿⣿⡟⠀⣼⣿⣿⣿⣦⣄⠙⠻⣿⣿⣿⣿⣿⣿⣿⠀⢻⣷⣦⣈⠙⠻⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡿⠿⠛⠛⠛⠿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿
// ⣿⣿⣿⣿⣿⠃⢰⣿⣿⣿⣿⣿⣿⣿⣦⡍⠙⠉⣁⣠⣤⣤⣄⡀⢻⣿⣿⣿⣦⣄⣈⠙⠿⢿⣿⣿⣿⣿⣿⣿⣿⡿⠟⠋⣀⣠⣴⣶⣿⣷⡄⠘⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿
// ⣿⣿⣿⣿⡏⠀⣼⣿⣿⣿⣿⣿⣿⣿⣿⣄⣀⠛⢿⣿⣿⣿⣿⣷⣾⣿⣿⣿⣿⣿⣿⣷⣶⣄⠛⣿⣿⣿⡿⠟⠋⣠⣴⣾⣿⣿⣿⣿⣿⣿⡇⠀⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿
// ⣿⣿⣿⣿⡇⢰⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣦⣌⠻⢿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣄⠘⣿⠋⠀⣴⣾⣿⣿⣿⣿⣿⣿⣿⣿⣿⡏⢠⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿
// ⣿⣿⣿⣿⠁⣾⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣶⣦⡄⠉⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣷⣤⣦⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⠀⢸⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿
// ⣿⣿⣿⣿⠀⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡏⠀⣼⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿
// ⣿⣿⣿⣿⠀⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡿⠁⣸⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿
// ⣿⣿⣿⣿⠀⢿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⠁⢠⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿
// ⣿⣿⣿⣿⡇⠘⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⢻⣿⡀⢻⣿⣿⣿⠏⢠⣾⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿
// ⣿⣿⣿⣿⣷⡀⠹⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡘⣿⠃⣸⣿⣿⠏⢀⣾⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿
// ⣿⡿⠿⠿⠛⠃⣠⣿⣿⡿⠟⠁⢀⣀⣀⡀⠉⠻⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣷⣶⣶⣿⡿⠋⢀⣼⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿
// ⣿⣷⡈⢶⣶⣿⣿⣿⣿⣦⣤⣾⣿⣿⣿⣿⣷⣀⢘⣿⣿⣿⣿⣿⣿⣿⣿⡿⠛⠉⠀⣀⣀⣀⠀⠉⠻⣿⣿⣿⣿⣿⣿⠟⠀⠀⠛⠛⢻⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿
// ⣿⣿⣷⣄⡛⠟⢿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣆⣠⣶⣿⣿⣿⣿⣿⣷⣄⠈⣿⣿⣿⣿⣿⣶⣾⣿⡟⠁⣸⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿
// ⣿⣿⣿⣿⡟⢁⣾⡟⠿⠛⠉⢻⣿⣿⣿⣿⣧⣀⡀⠀⠀⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⠿⠟⠁⣠⣾⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿
// ⣿⣿⣿⡿⠀⣿⣿⣿⣿⣿⡁⣉⣁⣤⣼⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡿⠿⠛⠛⠛⢿⡿⠿⢿⣿⣿⡀⠠⣴⣾⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿
// ⣿⣿⡟⠀⠚⠛⣉⣉⣉⡉⠛⢿⣿⣿⣿⣿⣿⣿⡿⢿⣿⠿⢿⣿⣿⡏⣿⣿⣿⣿⣿⣧⣴⣶⣧⡀⢉⣠⣶⣿⣿⣿⣷⡀⢿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿
// ⣿⣿⣷⣶⣿⣿⣿⣿⣿⣷⣦⡀⠙⠻⢿⣿⣿⣿⣧⣌⠉⣠⣬⣍⠋⢁⣿⣿⣿⣿⣿⣿⣿⣿⣿⣷⣾⣿⠿⢿⣿⣿⣿⡇⠘⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿
// ⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣶⣤⣀⣉⠙⠛⠿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡿⠿⠿⠟⠛⠋⠉⠠⠤⣤⣴⣶⣦⣤⣤⣄⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿
// ⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣦⡀⠠⣤⣤⣤⣤⣼⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡿⠿⠿⣿⣿⣿⠃⢀⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿
// ⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣦⡀⠉⢻⣿⣿⣿⣿⣿⣿⣿⣿⣿⠿⠋⣉⣠⣤⣶⣶⣤⣤⣄⠀⠸⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿
// ⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡟⠀⣸⣿⣿⣿⣿⣿⣿⣿⣿⠛⢁⣴⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣆⡈⢿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿
// ⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡿⠁⣴⣿⣿⣿⣿⣿⣿⣿⡟⢁⣾⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣷⡀⢹⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡟⠻⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿
// ⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡇⣀⣠⣤⡄⠸⣿⣿⣿⣿⣇⠸⣿⡏⢹⣿⣿⡿⢿⣿⣿⣿⣿⣿⣿⣿⣿⣧⡄⠹⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡇⠀⠈⢿⣿⣿⣿⣿⣿⣿⣿⣿
// ⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣷⠀⣿⣿⣿⣿⣿⣦⣈⠁⠘⠿⣿⡇⢸⠿⠟⢉⣠⣿⣿⣿⣿⣿⣷⡀⢻⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡇⠀⣧⡄⠹⣿⣿⣿⣿⣿⣿⣿
// ⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⠀⣿⣿⣿⣿⣿⣿⣿⣿⣶⣦⣤⣤⣤⡆⣿⣿⣿⣿⣿⣿⣿⣿⣿⣇⠈⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡿⠁⠹⣿⠃⢰⣿⣷⣄⠘⣿⣿⣿⣿⣿⣿
// ⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⠀⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡇⣿⣿⣿⣿⣿⣿⣿⣿⣿⡟⠀⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡿⠀⣰⡀⠈⠀⣿⣿⣿⣿⣄⠈⢻⣿⣿⣿⣿
// ⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⠀⣿⣿⣿⣿⣿⣿⣿⡏⢻⣿⣿⣿⣿⣇⠹⣿⣿⣿⣿⣿⣿⣿⡿⠁⣸⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡟⠁⣰⣿⣇⠀⢰⣿⣿⣿⣿⣿⣇⠈⢿⣿⣿⣿
// ⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⠀⣿⣿⣿⣿⣿⣿⣿⣧⠘⣿⣿⣿⣿⣿⣄⠻⣿⣿⣿⣿⡿⠟⢀⠰⠻⠿⠿⣿⣿⣿⣿⣿⣿⣿⡟⢀⣼⣿⣿⣿⢠⣿⣿⣿⣿⣿⣿⣿⣇⠈⢻⣿⣿
// ⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⠿⠛⠀⣿⣿⣿⣿⣿⣿⣿⣿⡄⢹⣿⣿⣿⣿⣿⣶⣤⣤⣤⣤⣴⣾⣿⣶⡶⠂⣴⣿⣿⣿⣿⡿⠟⠉⣠⣾⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡆⠈⣿⣿
// ⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡿⠋⢀⣰⣶⠀⣿⣿⣿⣿⣿⣿⣿⣿⣿⢸⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣀⠘⠿⢿⠿⠛⠁⣀⣴⣾⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣇⠀⣿⣿
// ⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡿⠁⣠⣿⣿⣿⠀⣿⣿⣿⣿⣿⣿⣿⣿⣿⠀⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⠛⢁⣠⣤⣤⣶⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⠀⢸⣿
// ⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⠀⢸⣿⣿⣿⣿⡀⢿⣿⣿⣿⣿⣿⣿⣿⣿⡀⢿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣷⠘⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⡟⠀⣼⣿
// ⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⠀⢼⣿⣿⣿⣿⡇⠘⣿⣿⣿⣿⣿⣿⣿⣿⡇⢸⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣆⠙⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⣿⠿⠁⣴⣿⣿